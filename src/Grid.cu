#include "Grid.h"
#include "utils.h"

hipError_t grid_allocate_gpu(struct grid_gpu* gpu_grd, size_t grd_arrays_size)
{
    hipError_t deviceError;
    deviceError =
        hipMalloc(&gpu_grd->XN_flat, sizeof(FPfield) * grd_arrays_size);
    if(deviceError != hipSuccess){
        printCudaError(deviceError);
        return deviceError;
    }

    deviceError =
        hipMalloc(&gpu_grd->YN_flat, sizeof(FPfield) * grd_arrays_size);
    if(deviceError != hipSuccess){
        printCudaError(deviceError);
        return deviceError;
    }

    deviceError =
        hipMalloc(&gpu_grd->ZN_flat, sizeof(FPfield) * grd_arrays_size);
    if(deviceError != hipSuccess){
        printCudaError(deviceError);
        return deviceError;
    }

    return hipSuccess;
}

void grid_deallocate_gpu(struct grid_gpu* gpu_grd) {
    hipFree(gpu_grd->XN_flat);
    hipFree(gpu_grd->YN_flat);
    hipFree(gpu_grd->ZN_flat);
}

void grid_cpy(struct grid_gpu* dst, struct grid* src)
{
    dst->nyn = src->nyn;
    dst->nzn = src->nzn;
    dst->invdx = src->invdx;
    dst->invdy = src->invdy;
    dst->invdz = src->invdz;
    dst->xStart = src->xStart;
    dst->yStart = src->yStart;
    dst->zStart = src->zStart;

    dst->invVOL = src->invVOL;

    dst->Lx = src->Lx;
    dst->Ly = src->Ly;
    dst->Lz = src->Lz;
}

void grid_cpy_to_gpu(struct grid_gpu* dst, struct grid* src)
{
    hipError_t deviceError;
    long grd_arrays_size = src->nxn * src->nyn * src->nzn;

    deviceError = hipMemcpy(dst->XN_flat, src->XN_flat,
                              sizeof(FPfield) * grd_arrays_size,
                              hipMemcpyHostToDevice);
    if(deviceError != hipSuccess){
        printCudaError(deviceError);
    }

    deviceError = hipMemcpy(dst->YN_flat, src->YN_flat,
                              sizeof(FPfield) * grd_arrays_size,
                              hipMemcpyHostToDevice);
    if(deviceError != hipSuccess){
        printCudaError(deviceError);
    }

    deviceError = hipMemcpy(dst->ZN_flat, src->ZN_flat,
                              sizeof(FPfield) * grd_arrays_size, hipMemcpyHostToDevice);
    if(deviceError != hipSuccess){
        printCudaError(deviceError);
    }
}

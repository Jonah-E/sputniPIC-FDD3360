#include "EMfield.h"
#include "utils.h"
#include <hip/hip_runtime.h>

hipError_t emfield_allocate_gpu(struct EMfield_gpu* gpu_field,
                                 size_t grd_arrays_size)
{
    hipError_t deviceError;
    deviceError =
        hipMalloc(&gpu_field->Ex_flat, sizeof(FPfield) * grd_arrays_size);
    if (deviceError != hipSuccess) {
        printCudaError(deviceError);
        return deviceError;
    }
    deviceError =
        hipMalloc(&gpu_field->Ey_flat, sizeof(FPfield) * grd_arrays_size);
    if (deviceError != hipSuccess) {
        printCudaError(deviceError);
        return deviceError;
    }
    deviceError =
        hipMalloc(&gpu_field->Ez_flat, sizeof(FPfield) * grd_arrays_size);
    if (deviceError != hipSuccess) {
        printCudaError(deviceError);
        return deviceError;
    }
    deviceError =
        hipMalloc(&gpu_field->Bxn_flat, sizeof(FPfield) * grd_arrays_size);
    if (deviceError != hipSuccess) {
        printCudaError(deviceError);
        return deviceError;
    }
    deviceError =
        hipMalloc(&gpu_field->Byn_flat, sizeof(FPfield) * grd_arrays_size);
    if (deviceError != hipSuccess) {
        printCudaError(deviceError);
        return deviceError;
    }
    deviceError =
        hipMalloc(&gpu_field->Bzn_flat, sizeof(FPfield) * grd_arrays_size);
    if (deviceError != hipSuccess) {
        printCudaError(deviceError);
        return deviceError;
    }

    return hipSuccess;
}

void emfield_deallocate_gpu(struct EMfield_gpu* gpu_field)
{
    hipFree(gpu_field->Ex_flat);
    hipFree(gpu_field->Ey_flat);
    hipFree(gpu_field->Ez_flat);

    hipFree(gpu_field->Bxn_flat);
    hipFree(gpu_field->Byn_flat);
    hipFree(gpu_field->Bzn_flat);
}

void emfield_cpy_to_gpu(struct EMfield_gpu* dst, struct EMfield* src,
                      size_t grd_arrays_size)
{
    hipError_t deviceError;
    deviceError =
        hipMemcpy(dst->Ex_flat, src->Ex_flat,
                   sizeof(FPfield) * grd_arrays_size, hipMemcpyHostToDevice);
    if (deviceError != hipSuccess) {
        printCudaError(deviceError);
    }
    deviceError =
        hipMemcpy(dst->Ey_flat, src->Ey_flat,
                   sizeof(FPfield) * grd_arrays_size, hipMemcpyHostToDevice);
    if (deviceError != hipSuccess) {
        printCudaError(deviceError);
    }
    deviceError =
        hipMemcpy(dst->Ez_flat, src->Ez_flat,
                   sizeof(FPfield) * grd_arrays_size, hipMemcpyHostToDevice);
    if (deviceError != hipSuccess) {
        printCudaError(deviceError);
    }

    deviceError =
        hipMemcpy(dst->Bxn_flat, src->Bxn_flat,
                   sizeof(FPfield) * grd_arrays_size, hipMemcpyHostToDevice);
    if (deviceError != hipSuccess) {
        printCudaError(deviceError);
    }
    deviceError =
        hipMemcpy(dst->Byn_flat, src->Byn_flat,
                   sizeof(FPfield) * grd_arrays_size, hipMemcpyHostToDevice);
    if (deviceError != hipSuccess) {
        printCudaError(deviceError);
    }
    deviceError =
        hipMemcpy(dst->Bzn_flat, src->Bzn_flat,
                   sizeof(FPfield) * grd_arrays_size, hipMemcpyHostToDevice);
    if (deviceError != hipSuccess) {
        printCudaError(deviceError);
    }
}
